#pragma comment(lib,"cublas.lib")

#include "hip/hip_runtime.h"
#include ""
#include "hipblas.h"

#include "cuda_header.cuh"

#include "Timer.h"

FLOAT_MAT_TYPE* cublas_mul_matrix(long aWidth, long aHeight, FLOAT_MAT_TYPE* aData, long bWidth, long bHeight, FLOAT_MAT_TYPE* bData){
	FLOAT_MAT_TYPE* matrix_a_dev;
	FLOAT_MAT_TYPE* matrix_b_dev;
	FLOAT_MAT_TYPE* matrix_out;
	FLOAT_MAT_TYPE* res;

	long width_out, height_out;
	width_out = bWidth;
	height_out = aHeight;

	Timer alloc_dev, exec, alloc_host;

	alloc_dev.start();
	hipMalloc((void**)&matrix_a_dev, aWidth * aHeight * sizeof(FLOAT_MAT_TYPE));
	hipMalloc((void**)&matrix_b_dev, bWidth * bHeight * sizeof(FLOAT_MAT_TYPE));
	hipMalloc((void**)&matrix_out, width_out * height_out * sizeof(FLOAT_MAT_TYPE));

	hipMemcpy(matrix_a_dev, aData, aWidth * aHeight * sizeof(FLOAT_MAT_TYPE), hipMemcpyHostToDevice);
	hipMemcpy(matrix_b_dev, bData, bWidth * bHeight * sizeof(FLOAT_MAT_TYPE), hipMemcpyHostToDevice);
	alloc_dev.stop("Allocate and copy to device: ");

	const float alpha = 1.0f;
	const float beta = 0.0f;
	hipblasHandle_t handle;

	exec.start();
	hipblasCreate(&handle);
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, bWidth, aHeight, aWidth, &alpha, matrix_b_dev, bWidth, matrix_a_dev, aWidth, &beta, matrix_out, bWidth);
	hipDeviceSynchronize();
	exec.stop("Execution: ");

	alloc_host.start();
	// copy result from device to host
	res = new FLOAT_MAT_TYPE[width_out * height_out];
	hipMemcpy(res, matrix_out, width_out * height_out * sizeof(FLOAT_MAT_TYPE), hipMemcpyDeviceToHost);

	hipblasDestroy(handle);
	alloc_host.stop("Allocate and copy to host: ");

	hipFree(matrix_a_dev);
	hipFree(matrix_b_dev);
	hipFree(matrix_out);

	return res;
}